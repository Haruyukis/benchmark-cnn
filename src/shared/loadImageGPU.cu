#include "hip/hip_runtime.h"
// #define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#include <hip/hip_complex.h>
#include "loadImageGPU.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

__global__ void kernelLoadImageGPU(unsigned char* imgCharDevice, hipFloatComplex* imgFloatDevice, 
                                   int width, int height, int channels, int trueWidth, int trueHeight) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    extern __shared__ unsigned char sharedMem[];

    int paddedIndex = y * width + x;
    int trueIndex = y * trueWidth + x;
    
    int threadId = threadIdx.y * blockDim.x + threadIdx.x;
    int sharedOffset = threadId * channels;

    if (x < trueWidth && y < trueHeight) {
        for (int c = 0; c < channels; c++) {
            sharedMem[sharedOffset + c] = imgCharDevice[trueIndex * channels + c];
        }
    } else {
        for (int c = 0; c < channels; c++) {
            sharedMem[sharedOffset + c] = 0;  // Zero padding
        }
    }

    __syncthreads();
    if (x < width && y < height) {
        for (int c = 0; c < channels; c++) {
            imgFloatDevice[c * width * height + paddedIndex] = 
                make_hipFloatComplex((float)sharedMem[sharedOffset + c], 0.0f);
        }
    }
}


void afficheImageFloat(hipFloatComplex* imgFloat, int width, int height, int channels){
    for (int i = 0; i<height*width; i++){
        if (i%width == 0){
            printf("\n");
        }
        printf("%3.0f",hipCrealf(imgFloat[i]));
    }
    printf("\n");
}

hipFloatComplex* loadImageGPU(const char* path, int* trueWidth, int* trueHeight, int* width, int* height, int* channels ){
    unsigned char* imgCharHost = stbi_load(path, trueWidth, trueHeight, channels,0);
    *width = 1<<(int) log2(*trueWidth-1)+1;
    *height = 1<<(int) log2(*trueHeight-1)+1;
    printf("Image chargée, width:%d, height:%d, channels:%d\n",*trueWidth, *trueHeight, *channels);

    // imgCharDevice
    unsigned char* imgCharDevice;
    hipMalloc(&imgCharDevice, (*channels)*(*width)*(*height)*sizeof(unsigned char));
    hipMemcpy(imgCharDevice, imgCharHost, (*channels)*(*width)*(*height)*sizeof(unsigned char), hipMemcpyHostToDevice);

    // imgFloatHost
    hipFloatComplex* imgFloatHost = (hipFloatComplex*) malloc((*channels)*(*width)*(*height)*sizeof(hipFloatComplex));

    // imgFloatDevice
    hipFloatComplex* imgFloatDevice;
    hipMalloc(&imgFloatDevice, (*channels)*(*width)*(*height)*sizeof(hipFloatComplex));

    int BLOCK_SIZE = 16;
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((*width + blockSize.x - 1) / blockSize.x, (*height + blockSize.y - 1) / blockSize.y);

    int sharedMemorySize = BLOCK_SIZE * BLOCK_SIZE * *channels * sizeof(unsigned char);
    
    kernelLoadImageGPU<<<gridSize, blockSize, sharedMemorySize>>>(imgCharDevice, imgFloatDevice, 
                                                                  *width, *height, *channels, 
                                                                  *trueWidth, *trueHeight);
    hipDeviceSynchronize();
    hipMemcpy(imgFloatHost, imgFloatDevice, (*width)*(*height)*(*channels)*sizeof(sizeof(hipFloatComplex)), hipMemcpyDeviceToHost);

    printf("Image paddée, width:%d, height:%d, channels:%d\n",*width, *height, *channels);
    return imgFloatDevice;
}

// int main(){
//     const char* path = "./data/gris.jpg";
//     int trueWidth, trueHeight, width, height, channels;
//     hipFloatComplex* sortie = loadImageCPU(path, &trueWidth, &trueHeight, &width, &height, &channels);
//     afficheImageFloat(sortie, width, height, channels);
//     return 0;
// }