#include "hip/hip_runtime.h"
// #define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#include "loadImageGPU.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

__global__ void kernelLoadImageGPU(unsigned char* imgCharDevice, hipFloatComplex* imgFloatDevice, 
                                   int width, int height, int channels, int trueWidth, int trueHeight) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    extern __shared__ unsigned char sharedMem[];

    int paddedIndex = y * width + x;
    int trueIndex = y * trueWidth + x;
    
    int threadId = threadIdx.y * blockDim.x + threadIdx.x;
    int sharedOffset = threadId * channels;

    if (x < trueWidth && y < trueHeight) {
        for (int c = 0; c < channels; c++) {
            sharedMem[sharedOffset + c] = imgCharDevice[trueIndex * channels + c];
        }
    } else {
        for (int c = 0; c < channels; c++) {
            sharedMem[sharedOffset + c] = 0;  // Zero padding
        }
    }

    __syncthreads();
    if (x < width && y < height) {
        for (int c = 0; c < channels; c++) {
            imgFloatDevice[c * width * height + paddedIndex] = 
                make_hipFloatComplex((float)sharedMem[sharedOffset + c], 0.0f);
        }
    }
}

__global__ void kernelLoadImageGPUf(unsigned char* imgCharDevice, float* imgFloatDevice, 
                                   int width, int height, int channels, int trueWidth, int trueHeight) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    extern __shared__ unsigned char sharedMem[];

    int paddedIndex = y * width + x;
    int trueIndex = y * trueWidth + x;
    
    int threadId = threadIdx.y * blockDim.x + threadIdx.x;
    int sharedOffset = threadId * channels;

    if (x < trueWidth && y < trueHeight) {
        for (int c = 0; c < channels; c++) {
            sharedMem[sharedOffset + c] = imgCharDevice[trueIndex * channels + c];
        }
    } else {
        for (int c = 0; c < channels; c++) {
            sharedMem[sharedOffset + c] = 0;  // Zero padding
        }
    }

    __syncthreads();
    if (x < width && y < height) {
        for (int c = 0; c < channels; c++) {
            imgFloatDevice[c * width * height + paddedIndex] = (float) sharedMem[sharedOffset + c];
        }
    }
}

float* loadImageGPUf(const char* path, int* trueWidth, int* trueHeight, int* width, int* height, int* channels ){
    unsigned char* imgCharHost = stbi_load(path, trueWidth, trueHeight, channels,0);
    *width = 1<<(int) log2(*trueWidth-1)+1;
    *height = 1<<(int) log2(*trueHeight-1)+1;
    printf("Image chargée, width:%d, height:%d, channels:%d\n",*trueWidth, *trueHeight, *channels);

    // imgCharDevice
    unsigned char* imgCharDevice;
    hipMalloc(&imgCharDevice, (*channels)*(*width)*(*height)*sizeof(unsigned char));
    hipMemcpy(imgCharDevice, imgCharHost, (*channels)*(*width)*(*height)*sizeof(unsigned char), hipMemcpyHostToDevice);

    // imgFloatHost
    float* imgFloatHost = (float*) malloc((*channels)*(*width)*(*height)*sizeof(float));

    // imgFloatDevice
    float* imgFloatDevice;
    hipMalloc(&imgFloatDevice, (*channels)*(*width)*(*height)*sizeof(float));

    int BLOCK_SIZE = 16;
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((*width + blockSize.x - 1) / blockSize.x, (*height + blockSize.y - 1) / blockSize.y);

    int sharedMemorySize = BLOCK_SIZE * BLOCK_SIZE * *channels * sizeof(unsigned char);
    
    kernelLoadImageGPUf<<<gridSize, blockSize, sharedMemorySize>>>(imgCharDevice, imgFloatDevice, 
                                                                  *width, *height, *channels, 
                                                                  *trueWidth, *trueHeight);
    hipDeviceSynchronize();
    printf("Image paddée, width:%d, height:%d, channels:%d\n",*width, *height, *channels);
    
    // Clean 
    stbi_image_free(imgCharHost);
    hipFree(imgCharDevice);
    free(imgFloatHost);
    return imgFloatDevice;
}

hipFloatComplex* loadImageGPU(const char* path, int* trueWidth, int* trueHeight, int* width, int* height, int* channels ){
    unsigned char* imgCharHost = stbi_load(path, trueWidth, trueHeight, channels,0);
    *width = 1<<(int) log2(*trueWidth-1)+1;
    *height = 1<<(int) log2(*trueHeight-1)+1;
    printf("Image chargée, width:%d, height:%d, channels:%d\n",*trueWidth, *trueHeight, *channels);

    // imgCharDevice
    unsigned char* imgCharDevice;
    hipMalloc(&imgCharDevice, (*channels)*(*width)*(*height)*sizeof(unsigned char));
    hipMemcpy(imgCharDevice, imgCharHost, (*channels)*(*width)*(*height)*sizeof(unsigned char), hipMemcpyHostToDevice);

    // imgFloatHost
    hipFloatComplex* imgFloatHost = (hipFloatComplex*) malloc((*channels)*(*width)*(*height)*sizeof(hipFloatComplex));

    // imgFloatDevice
    hipFloatComplex* imgFloatDevice;
    hipMalloc(&imgFloatDevice, (*channels)*(*width)*(*height)*sizeof(hipFloatComplex));

    int BLOCK_SIZE = 16;
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((*width + blockSize.x - 1) / blockSize.x, (*height + blockSize.y - 1) / blockSize.y);

    int sharedMemorySize = BLOCK_SIZE * BLOCK_SIZE * *channels * sizeof(unsigned char);
    
    kernelLoadImageGPU<<<gridSize, blockSize, sharedMemorySize>>>(imgCharDevice, imgFloatDevice, 
                                                                  *width, *height, *channels, 
                                                                  *trueWidth, *trueHeight);
    hipDeviceSynchronize();
    printf("Image paddée, width:%d, height:%d, channels:%d\n",*width, *height, *channels);
    
    // Clean 
    stbi_image_free(imgCharHost);
    hipFree(imgCharDevice);
    free(imgFloatHost);
    return imgFloatDevice;
}

