#include "hip/hip_runtime.h"
#include "hadamard.cuh"

__global__ void hadamard_kernel(float* C, const float* A, const float* B, unsigned int width, unsigned int height) {
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned int index = row * width + col;

    if (row < height && col < width) {
        C[index] = A[index] * B[index];
    }
}

void hadamard(float* C, const float* A, const float* B, int width, int height) {
    float *d_A, *d_B, *d_C;
    size_t size = width * height * sizeof(float);

    hipMalloc((void **) &d_A, size);
    hipMalloc((void **) &d_B, size);
    hipMalloc((void **) &d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 blockDim(32, 32);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
    
    hadamard_kernel<<<gridDim, blockDim>>>(d_C, d_A, d_B, width, height);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}