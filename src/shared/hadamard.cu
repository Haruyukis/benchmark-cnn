#include "hip/hip_runtime.h"
#include "hadamard.cuh"

__global__ void hadamard_kernel(MatrixVo C, const MatrixVo A, const MatrixVo B, unsigned int width, unsigned int height) {
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned int index = row * width + col;

    if (row < height && col < width) {
        C[index] = A[index] * B[index];
    }
}

void hadamard(double* C, const double* A, const double* B, int width, int height) {
    double *d_A, *d_B, *d_C;
    size_t size = width * height * sizeof(double);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 blockDim(32, 32);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
    
    hadamard_kernel<<<gridDim, blockDim>>>(d_C, d_A, d_B, width, height);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}