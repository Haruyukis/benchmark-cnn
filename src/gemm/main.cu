#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#include <iostream>
#include <hip/hip_runtime.h>
#include "../shared/gemm.cuh"
#include "../shared/loadImageGPU.cu"
#include "../shared/storeImageGPU.cu"

using namespace std;

// Simple error-checking macro.
#define CUDA_CHECK(call)                                                      \
    {                                                                         \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                             \
            std::cerr << "CUDA error in " << __FILE__ << " at line "          \
                      << __LINE__ << ": " << hipGetErrorString(err) << "\n";  \
            exit(err);                                                        \
        }                                                                     \
    }

//---------------------------------------------------------------------
// im2col_kernel: transforms the input image into a matrix where each 
// column is one 3x3 patch (across 3 channels) from the image.
// The input image is assumed to be stored in channels-first order
// (i.e., [channel][row][col]).
//---------------------------------------------------------------------
__global__ void im2col_kernel(float *image, float *cols,
                              int channels, int height, int width,
                              int kH, int kW, int outH, int outW)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int numPatches = outH * outW;
    if (idx < numPatches) {
        // Compute the top-left position of the patch.
        int w_out = idx % outW;
        int h_out = idx / outW;
        // For each channel and kernel element.
        for (int c = 0; c < channels; c++) {
            for (int i = 0; i < kH; i++) {
                for (int j = 0; j < kW; j++) {
                    int row = c * kH * kW + i * kW + j;
                    int in_row = h_out + i;
                    int in_col = w_out + j;
                    cols[row * numPatches + idx] =
                        image[c * height * width + in_row * width + in_col];
                }
            }
        }
    }
}

//---------------------------------------------------------------------
// gemm: A simple matrix multiplication kernel that computes:
//        C = A * B
// where A is (M x K), B is (K x N), and C is (M x N).
// In our convolution, A is the flattened kernel (size 1x27),
// B is the im2col matrix (27 x numPatches),
// and C is (1 x numPatches).
//---------------------------------------------------------------------
__global__ void gemm_debug(float *A, float *B, float *C,
                     int M, int N, int K)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y; // M dimension
    int col = blockIdx.x * blockDim.x + threadIdx.x; // N dimension
    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; k++) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

//---------------------------------------------------------------------
// conv_im2col_gemm: Host function to perform convolution using the
// im2col transformation and GEMM.
//  - d_image_in:  device pointer to input image (3, height, width)
//  - d_kernel:    device pointer to the kernel (flattened 27 floats)
//  - d_image_out: device pointer to output (size: (height-2) x (width-2))
//  - height, width: dimensions of the input image
//---------------------------------------------------------------------
void conv_im2col_gemm(float *d_image_in, float *d_kernel,
                      float *d_image_out, int height, int width)
{
    const int channels = 3;
    const int kH = 3, kW = 3;
    int outH = height - kH + 1;
    int outW = width - kW + 1;
    int numPatches = outH * outW;
    int kernelSize = channels * kH * kW; // 27

    // Allocate device memory for the im2col matrix.
    float *d_cols = nullptr;
    CUDA_CHECK(hipMalloc(&d_cols, kernelSize * numPatches * sizeof(float)));

    // Launch im2col kernel.
    int threadsPerBlock = 256;
    int numBlocks = (numPatches + threadsPerBlock - 1) / threadsPerBlock;
    im2col_kernel<<<numBlocks, threadsPerBlock>>>(d_image_in, d_cols,
                                                  channels, height, width,
                                                  kH, kW, outH, outW);
    CUDA_CHECK(hipDeviceSynchronize());

    // Perform GEMM: (1 x 27) * (27 x numPatches) = (1 x numPatches).
    // We use a 2D thread grid.
    int gemmBlockSize = 16;
    gemm(d_kernel, d_cols, d_image_out, kernelSize, 1, numPatches, gemmBlockSize);
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipFree(d_cols));
}

//---------------------------------------------------------------------
// Main function:
//   - Loads "poupoupidou.jpg" using OpenCV.
//   - Converts the image to float and rearranges it from HWC (OpenCV)
//     to CHW order (required by the CUDA kernels).
//   - Sets up a 3x3x3 averaging kernel.
//   - Copies data to the GPU, runs the convolution, and copies the
//     result back.
//   - Saves the output as "output.jpg".
//---------------------------------------------------------------------
int main(int argc, char *argv[])
{
    // Load the image from file.
    if (argc != 2){
        fprintf(stderr, "Usage: %s <chemin_image>\n", argv[0]);
        return EXIT_FAILURE;
    }

    const char* path = argv[1];
    int width, height, channels;
    float* d_image_in = loadImageGPUfGemm(path, &width, &height, &channels);

    std::cout << "Loaded image: " << path << " with size " 
              << width << "x" << height << std::endl;

    if (channels != 3) {
        std::cerr << "Error: Image must have 3 channels." << std::endl;
        return -1;
    }

    // Define kernel.
    const int kernelSize = 3 * 3 * 3; // 27 elements.
    float h_kernel[kernelSize];
    for (int i = 0; i < 3; i++) {
        h_kernel[i] = -1;
        h_kernel[i+9] = -1;
        h_kernel[i+18] = -1;
    }
    for (int i = 3; i < 6; i++) {
        h_kernel[i] = 0;
        h_kernel[i+9] = 0;
        h_kernel[i+18] = 0;
    }
    for (int i = 6; i < 9; i++) {
        h_kernel[i] = 1;
        h_kernel[i+9] = 1;
        h_kernel[i+18] = 1;
    }

    // Allocate device memory for the kernel and copy it.
    float *d_kernel = nullptr;
    CUDA_CHECK(hipMalloc(&d_kernel, kernelSize * sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_kernel, h_kernel, kernelSize * sizeof(float),
                            hipMemcpyHostToDevice));

    // Determine the output dimensions.
    // With a 3x3 convolution (no padding, stride=1), the output is (height-2)x(width-2).
    int outH = height - 2;
    int outW = width - 2;
    size_t outputSize = outH * outW;

    // Allocate device memory for the output.
    float *d_image_out = nullptr;
    CUDA_CHECK(hipMalloc(&d_image_out, outputSize * sizeof(float)));

    // Run the convolution.
    conv_im2col_gemm(d_image_in, d_kernel, d_image_out, height, width);

    // // Copy the output back to host.
    // float *h_output = new float[outputSize];
    // CUDA_CHECK(hipMemcpy(h_output, d_image_out, outputSize * sizeof(float),
    //                         hipMemcpyDeviceToHost));

    // // Convert output to displayable format
    // float a = 1024; 
    // float b = -1024;
    // float pixel;
    // for (int i = 0; i < outW * outH; i++) {
    //     pixel = h_output[i];
    //     if (pixel < a) {
    //         a = pixel;
    //     } else if (pixel > b) {
    //         b = pixel;
    //     }
    // }

    // float div = b - a;
    // for (int i = 0; i < outW * outH; i++) {
    //     h_output[i] = 255 * (h_output[i] - a) / div;
    //     // printf("%f\n", h_output[i]);
    // }

    // CUDA_CHECK(hipMemcpy(d_image_out, h_output, outputSize * sizeof(float),
    //                         hipMemcpyHostToDevice));

    // Write the output image to file.
    const char* outFile = "../../out/output.jpg";
    storeImageGPUfGemm(d_image_out, outFile, outW, outH, 1);
    std::cout << "Output image saved as " << outFile << std::endl;

    // Free host and device memory.
    CUDA_CHECK(hipFree(d_image_in));
    CUDA_CHECK(hipFree(d_kernel));
    CUDA_CHECK(hipFree(d_image_out));

    return 0;
}
