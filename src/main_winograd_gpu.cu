#include "winograd/winograd.cuh"
#include "shared/loadImageGPU.cuh"
#include "shared/storeImageGPU.cuh"
#include <iostream>
#include <hip/hip_runtime.h>
#include "shared/utils.cpp"
#include "shared/storeImage.hpp"

int main(int argc, char *argv[]){
    if (argc != 2){
        fprintf(stderr, "Usage: %s <chemin_image>\n", argv[0]);
        return EXIT_FAILURE;
    }

    const char* path = argv[1];
    int width, height, nb_channels;
    float* d_input = loadImageGPUf(path, &width, &height, &nb_channels);
    float* d_output;
    
    int o_width = width - 2;
    int o_height = height - 2;
    int i_size = width * height;
    int o_size = o_width * o_height;
    float* filter = new float[9]{
        -1.0f, -1.f, -1.f,
        0.0f, 0.f, 0.f,
        1.0f, 1.f, 1.f
    };
    
    hipMalloc((void **) &d_output, o_size * sizeof(float) * nb_channels);
    winograd_host(d_output, d_input, filter, width, height, 3, 3, nb_channels, 1);

    storeImageGPUf(d_output, "output_gpu.jpg", o_width, o_height, nb_channels);
    
    return 0;
}