#include "hip/hip_runtime.h"
#include "fft_shared_row.cu"
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include "../shared/transpose.cuh"
#include "../shared/loadImage.hpp"
#include "../shared/storeImage.hpp"

void fftShared(hipFloatComplex** img_complexe, int width, int height, int channels){
    int N = width*height;
    int log2width = (int)log2(width);
    int log2height = (int)log2(height);
    // For each channel, do the FFT
    for (int channel = 0; channel<channels; channel++){
        hipFloatComplex *d_data;
        hipMalloc((void **)&d_data, N * sizeof(hipFloatComplex));
        // Copy input data from host to device
        hipMemcpy(d_data, img_complexe[channel], N * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

        dim3 threadsPerBlock(width);    // One thread per element in the row
        dim3 blocksPerGrid(height);     // One block per row
        // Allocate shared memory for each row
        int sharedMemorySize = width * sizeof(hipFloatComplex);
        // Launch FFT kernel for each row of the image
        fft_DIF_on_rows<<<blocksPerGrid, threadsPerBlock, sharedMemorySize>>>(d_data, width, height, log2width);

        // Wait for kernel to finish
        hipDeviceSynchronize();
        dim3 blockDim(32, 32);
        dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

        // Taille de la mémoire partagée
        size_t sharedMemSize = blockDim.x * (blockDim.y + 1) * sizeof(hipFloatComplex);

        hipFloatComplex* dataTransposed;
        hipMalloc((void**)&dataTransposed ,N*sizeof(hipFloatComplex));

        transposeCF<<<gridDim, blockDim, sharedMemSize>>>(d_data, dataTransposed, width, height);
        hipDeviceSynchronize();

        
        fft_DIF_on_rows<<<blocksPerGrid, threadsPerBlock, sharedMemorySize>>>(dataTransposed, height, width, log2height);
        hipDeviceSynchronize();
        
        transposeCF<<<gridDim, blockDim, sharedMemSize>>>(dataTransposed, d_data, height, width);
        hipDeviceSynchronize();
        hipFree(dataTransposed);
        // Copy the result back from device to host
        
        hipMemcpy(img_complexe[channel], d_data, N * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
        hipFree(d_data);
    }
}

// Main program
int main(){
    int width, height, channels;
    const char* chemin_image = "./data/Te-noTr_0000_padded.jpg";
    float** image = loadImageF(chemin_image, &width, &height, &channels);
    
    const int N = width * height; // Total number of elements (pixels in the image)
    const int log2width = (int)log2f(width);
    const int log2height = (int)log2f(height);

    // Fait le taf de la fonction load_complexe
    hipFloatComplex** img_complexe = (hipFloatComplex**)malloc(channels*sizeof(hipFloatComplex*));
    for (int channel = 0; channel < channels; channel++){
        img_complexe[channel] = (hipFloatComplex *)malloc(N * sizeof(hipFloatComplex));
        for (int n = 0; n < N; n++){
            img_complexe[channel][n] = make_hipFloatComplex(image[channel][n], 0);
        }
    }

    fftShared(img_complexe, width, height, channels);

    // Output the result
    for (int channel = 0; channel<channels; channel++){
        for (int i = 0; i < N; ++i) {
        image[channel][i] = hipCrealf(img_complexe[channel][i]);
        // printf("Output[%d] = (%.2f, %.2f)\n", i, hipCrealf(h_input[i]), hipCimagf(h_input[i]));
        }
    }
    const char* chemin_sortie = "./data/test fft.jpeg";
    storeImageF(chemin_sortie, image, width, height, channels);

    


    

    

    

    
    

    /*
    // REVERSE

    ifft_DIT_on_rows<<<blocksPerGrid, threadsPerBlock, sharedMemorySize>>>(d_data, width, height, log2width);

    // Wait for kernel to finish
    hipDeviceSynchronize();

    // Copy the result back from device to host
    hipMemcpy(h_input, d_data, N * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);

    // Output the result
    for (int i = 0; i < N; ++i) {
        image[0][i] = hipCrealf(h_input[i])/width;
        // printf("Output[%d] = (%.2f, %.2f)\n", i, hipCrealf(h_input[i]), hipCimagf(h_input[i]));
    }
    const char* chemin_sortie_ifft = "./data/test ifft.jpeg";
    storeImageF(chemin_sortie_ifft, image, width, height, channels);

    // Clean up memory
    hipFree(d_data);
    free(h_input);
    */

    return 0;
}

/*
nvcc -ccbin /usr/bin/gcc-10 src/fft/TestFFTRows.cu src/shared/loadImage.c src/shared/storeImage.c -o build/TestFFTRows -lm -g
*/
