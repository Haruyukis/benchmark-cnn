#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include "../shared/loadImage.hpp"
#include "../shared/storeImage.hpp"
#include "convFFTShared.cuh"



// Main program
int main(){
    int width, height, channels;
    const char* chemin_image = "./data/64.jpg";
    float** image = loadImageF(chemin_image, &width, &height, &channels);
    
    const int N = width * height; // Total number of elements (pixels in the image)
    const int log2width = (int)log2f(width-1)+1;
    const int log2height = (int)log2f(height-1)+1;
    printf("log2width : %d\n",log2width);
    printf("log2height : %d",log2height);
    // Fait le taf de la fonction load_complexe
    hipFloatComplex** img_complexe = (hipFloatComplex**)malloc(channels*sizeof(hipFloatComplex*));
    for (int channel = 0; channel < channels; channel++){
        img_complexe[channel] = (hipFloatComplex *)malloc(N * sizeof(hipFloatComplex));
        for (int n = 0; n < N; n++){
            img_complexe[channel][n] = make_hipFloatComplex(image[channel][n], 0);
        }
    }
    hipFloatComplex* imgDevice;
    hipMalloc(&imgDevice, channels * N * sizeof(hipFloatComplex));

    // Step 2: Allocate memory for each channel on the device
    for (int channel = 0; channel < channels; channel++) {
    hipFloatComplex* ptrChannel = imgDevice + channel * N;  // Correct pointer arithmetic

    // Assuming img_complexe[channel * N] is the start of the channel data on the host
    hipMemcpy(ptrChannel, img_complexe[channel], N * sizeof(hipFloatComplex), hipMemcpyHostToDevice);
    }
    
    // Output the result of the first FFT
    // for (int channel = 0; channel<channels; channel++){
    //     for (int i = 0; i < N; ++i) {
    //     image[channel][i] = hipCrealf(img_complexe[channel][i]);
    //     // printf("Output[%d] = (%.2f, %.2f)\n", i, hipCrealf(h_input[i]), hipCimagf(h_input[i]));
    //     }
    // }
    // const char* chemin_sortie = "./data/test fft_apres_vla_modifs.jpeg";
    // storeImageF(chemin_sortie, image, width, height, channels);

    hipFloatComplex*kernel_h = (hipFloatComplex*)calloc(N,sizeof(hipFloatComplex));
    kernel_h[0*width + 0] = make_hipFloatComplex(-1,0);
    kernel_h[0*width + 1] = make_hipFloatComplex(-1,0);
    kernel_h[0*width + 2] = make_hipFloatComplex(-1,0);

    kernel_h[1*width + 0] = make_hipFloatComplex(0,0);
    kernel_h[1*width + 1] = make_hipFloatComplex(0,0);
    kernel_h[1*width + 2] = make_hipFloatComplex(0,0);

    kernel_h[2*width + 0] = make_hipFloatComplex(1,0);
    kernel_h[2*width + 1] = make_hipFloatComplex(1,0);
    kernel_h[2*width + 2] = make_hipFloatComplex(1,0);
    hipFloatComplex* kernel_d;
    hipMalloc((void**)&kernel_d, N*sizeof(hipFloatComplex));
    hipMemcpy(kernel_d, kernel_h, N * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    convFFTShared(img_complexe, imgDevice, kernel_d, width, height, channels);
    
    // Output the result
    for (int channel = 0; channel<channels; channel++){
        for (int i = 0; i < N; ++i) {
        image[channel][i] = hipCrealf(img_complexe[channel][i]);
        // printf("Output[%d] = (%.2f, %.2f)\n", i, hipCrealf(h_input[i]), hipCimagf(h_input[i]));
        }
    }
    const char* chemin_sortie_inv = "./data/test 64_INVERSE?.jpeg";
    storeImageF(chemin_sortie_inv, image, width, height, channels);
    return 0;
}

/*
nvcc -ccbin /usr/bin/gcc-10 src/fft/TestFFTRows.cu src/shared/loadImage.c src/shared/storeImage.c -o build/TestFFTRows -lm -g
*/
