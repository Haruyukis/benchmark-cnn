#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include "../shared/loadImage.hpp"
#include "../shared/storeImage.hpp"
#include "convFFTShared.cuh"
#include "../shared/loadImageGPU.cuh"


// Main program
int main(){
    int trueWidth, trueHeight, width, height, channels;
    const char* path = "./data/Te-noTr_0000.jpg";
    hipFloatComplex* imgDevice = loadImageGPU(path, &trueWidth, &trueHeight, &width, &height, &channels);
    int N = width*height;

    hipFloatComplex*kernel_h = (hipFloatComplex*)calloc(N,sizeof(hipFloatComplex));
    kernel_h[0*width + 0] = make_hipFloatComplex(-1,0);
    kernel_h[0*width + 1] = make_hipFloatComplex(-1,0);
    kernel_h[0*width + 2] = make_hipFloatComplex(-1,0);

    kernel_h[1*width + 0] = make_hipFloatComplex(0,0);
    kernel_h[1*width + 1] = make_hipFloatComplex(0,0);
    kernel_h[1*width + 2] = make_hipFloatComplex(0,0);

    kernel_h[2*width + 0] = make_hipFloatComplex(1,0);
    kernel_h[2*width + 1] = make_hipFloatComplex(1,0);
    kernel_h[2*width + 2] = make_hipFloatComplex(1,0);

    hipFloatComplex* kernel_d;
    hipMalloc((void**)&kernel_d, N*sizeof(hipFloatComplex));
    hipMemcpy(kernel_d, kernel_h, N * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    convFFTShared(imgDevice, kernel_d, width, height, channels);

    // TAF de storeGPU
    hipFloatComplex** img_complexe = (hipFloatComplex**)malloc(channels*sizeof(hipFloatComplex*));
    for (int channel = 0; channel < channels; channel++){
        img_complexe[channel] = (hipFloatComplex *)malloc(N * sizeof(hipFloatComplex));
        hipFloatComplex* ptrChannel = imgDevice + channel * N;   // ptr to the channel
        hipMemcpy(img_complexe[channel], ptrChannel, N * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    }
    float** image = (float**)malloc(channels*sizeof(float*));
    for (int channel = 0; channel < channels; channel++){
        image[channel] = (float*)malloc(N*sizeof(float));
    }
    // Output the result
    for (int channel = 0; channel<channels; channel++){
        for (int i = 0; i < N; ++i) {
        image[channel][i] = hipCrealf(img_complexe[channel][i]);
        // printf("Output[%d] = (%.2f, %.2f)\n", i, hipCrealf(h_input[i]), hipCimagf(h_input[i]));
        }
    }
    const char* chemin_sortie_inv = "./data/convFFT.jpeg";
    storeImageF(chemin_sortie_inv, image, width, height, channels);
    // Clean :
    hipFree(imgDevice);
    for (int channel = 0; channel < channels; channel++){
        free(img_complexe[channel]);
        free(image[channel]);
    }
    free(image);
    free(img_complexe);
    free(kernel_h);
    hipFree(kernel_d);

    return 0;
}

/*
nvcc -ccbin /usr/bin/gcc-10 src/fft/TestFFTRows.cu src/shared/loadImage.c src/shared/storeImage.c -o build/TestFFTRows -lm -g
*/
