#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include "../shared/loadImage.hpp"
#include "../shared/storeImage.hpp"
#include "convFFTShared.cuh"
#include "../shared/loadImageGPU.cuh"
#include "../shared/storeImageGPU.cuh"

// Main program
int main(){
    int trueWidth, trueHeight, width, height, channels;
    const char* path = "./data/ensimag.jpg";
    hipFloatComplex* imgDevice = loadImageGPU(path, &trueWidth, &trueHeight, &width, &height, &channels);
    int N = width*height;

    hipFloatComplex*kernel_h = (hipFloatComplex*)calloc(N,sizeof(hipFloatComplex));
    kernel_h[0*width + 0] = make_hipFloatComplex(-1,0);
    kernel_h[0*width + 1] = make_hipFloatComplex(-1,0);
    kernel_h[0*width + 2] = make_hipFloatComplex(-1,0);

    kernel_h[1*width + 0] = make_hipFloatComplex(0,0);
    kernel_h[1*width + 1] = make_hipFloatComplex(0,0);
    kernel_h[1*width + 2] = make_hipFloatComplex(0,0);

    kernel_h[2*width + 0] = make_hipFloatComplex(1,0);
    kernel_h[2*width + 1] = make_hipFloatComplex(1,0);
    kernel_h[2*width + 2] = make_hipFloatComplex(1,0);

    hipFloatComplex* kernel_d;
    hipMalloc((void**)&kernel_d, N*sizeof(hipFloatComplex));
    hipMemcpy(kernel_d, kernel_h, N * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    convFFTShared(imgDevice, kernel_d, width, height, channels);

    const char* PAF = "./data/sortieStoreGPU.jpeg";
    
    storeImageGPU(imgDevice, PAF, trueWidth, trueHeight, width, height, channels);

    // Clean :
    hipFree(imgDevice);
    free(kernel_h);
    hipFree(kernel_d);

    return 0;
}

/*
nvcc -ccbin /usr/bin/gcc-10 src/fft/TestFFTRows.cu src/shared/loadImage.c src/shared/storeImage.c -o build/TestFFTRows -lm -g
*/
