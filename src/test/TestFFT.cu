#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include "fftSharedRow.cuh"

int main(){
    int N = 12;
    hipFloatComplex* testFFT_h = (hipFloatComplex*) malloc(N*sizeof(hipFloatComplex));
    hipFloatComplex* out = (hipFloatComplex*) malloc(N*sizeof(hipFloatComplex));
    for (int i = 0; i<N; i++){
        testFFT_h[i] = make_hipFloatComplex(cosf(2.0f * M_PI * i / N), sinf(2.0f * M_PI * i / N));
        printf("Input[%d] = (%.2f, %.2f)\n", i, hipCrealf(testFFT_h[i]), hipCimagf(testFFT_h[i]));
        out[i] = make_hipFloatComplex(0,0);
    }

    hipFloatComplex* testFFT_d;
    hipMalloc(&testFFT_d, N*sizeof(hipFloatComplex));
    hipMemcpy(testFFT_d, testFFT_h, N * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    // Design space for the FFT
    dim3 threadsPerBlock(2*N);    // One thread per element in the row
    dim3 blocksPerGrid(2*1);     // One block per row
    int sharedMemorySize = N * sizeof(hipFloatComplex) * 10; // Allocate shared memory for each row

    fft_DIF_on_rows<<<blocksPerGrid, threadsPerBlock, sharedMemorySize>>>(testFFT_d, N, 1, (int) log2(N));
    hipDeviceSynchronize();

    hipMemcpy(out, testFFT_d, N * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    printf("\n");
    for (int i = 0; i<N; i++){
        printf("Output[%d] = (%.2f, %.2f)\n", i, hipCrealf(out[i]), hipCimagf(out[i]));
    }
    hipFree(testFFT_d);
    free(testFFT_h);
    return 0;
}