#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include "../shared/loadImage.hpp"
#include "../shared/storeImage.hpp"
#include "convFFTShared.cuh"
#include "../shared/loadImageGPU.cuh"
#include "../shared/storeImageGPU.cuh"

// Main program
int main(int argc, char *argv[]){
    if (argc != 2){
        fprintf(stderr, "Usage: %s <chemin_image>\n", argv[0]);
        return EXIT_FAILURE;
    }

    const char* path = argv[1];
    int trueWidth, trueHeight, width, height, channels;
    hipFloatComplex* imgDevice = loadImageGPU(path, &trueWidth, &trueHeight, &width, &height, &channels);
    int N = width*height;

    hipFloatComplex*kernel_h = (hipFloatComplex*)calloc(N,sizeof(hipFloatComplex));
    kernel_h[0*width + 0] = make_hipFloatComplex(-1,0);
    kernel_h[0*width + 1] = make_hipFloatComplex(-1,0);
    kernel_h[0*width + 2] = make_hipFloatComplex(-1,0);

    kernel_h[1*width + 0] = make_hipFloatComplex(0,0);
    kernel_h[1*width + 1] = make_hipFloatComplex(0,0);
    kernel_h[1*width + 2] = make_hipFloatComplex(0,0);

    kernel_h[2*width + 0] = make_hipFloatComplex(1,0);
    kernel_h[2*width + 1] = make_hipFloatComplex(1,0);
    kernel_h[2*width + 2] = make_hipFloatComplex(1,0);

    hipFloatComplex* kernel_d;
    hipMalloc((void**)&kernel_d, N*sizeof(hipFloatComplex));
    hipMemcpy(kernel_d, kernel_h, N * sizeof(hipFloatComplex), hipMemcpyHostToDevice);

    convFFTShared(imgDevice, kernel_d, width, height, channels);

    const char* PAF = "output.jpeg";
    
    storeImageGPU(imgDevice, PAF, trueWidth, trueHeight, width, height, channels);

    // Clean :
    hipFree(imgDevice);
    free(kernel_h);
    hipFree(kernel_d);

    return 0;
}

/*
nvcc -ccbin /usr/bin/gcc-10 src/fft/TestFFTRows.cu src/shared/loadImage.c src/shared/storeImage.c -o build/TestFFTRows -lm -g
*/
